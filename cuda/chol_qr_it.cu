
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <cutil.h>
#include "hipblas.h"
//#include </opt/eecs/Matlab/R2015b/toolbox/rtw/rtwdemos/crl_demo/cblas.h>

//=============================================================================

extern "C" int sgemm_(char *, char *, int *, int *, int *, float *, float *, 
                      int *, float *, int *, float *, float *, int *);
extern "C" int sgesvd_(char *, char *, int *, int *, float *, int *, float *, 
                       float *, int *, float *, int *, float *, int *, int *);
extern "C" void sgeqrf_(int*, int*, float*, int*, float*, float*, int*, int*);
extern "C" int scopy_(int *, float*, int *, float*, int *);
extern "C" int strmm_(char*, char *, char*, char *, int *, int *, float *, 
                      float *, int *, float *, int *);
extern "C" int strsm_(char *, char *, char *, char *, int *, int *, 
                      float *, float *, int *, float *, int *);
extern "C" int ssyrk_(char *, char *, int *, int *, float *, float *, 
                      int *, float *, float *, int *);
//=============================================================================

void chol_qr_it(int m, int n, float *A, int lda, float *R){
    int i = 0, k, j, info, lwork = n*n, n2 = n*n, one = 1;
    float *G, *U, *S, *VT, *vt, *tau, *work;  
    float cn = 200.f, alpha = 1.f, zero = 0.f, mins, maxs;
    
    G    = (float*)malloc(n * n * 4);
    VT   = (float*)malloc(n * n * 4);
    S    = (float*)malloc(    n * 4);
    work = (float*)malloc(lwork * 4);
    tau  = (float*)malloc(    n * 4);  

    do {
      i++;

      sgemm_("t", "n", &n, &n, &m, &alpha, A, &m, A, &m, &zero, G, &n);
      //ssyrk_('l', 't', &n, &m, &alpha, A, &m, &zero, G, &n);
      //for(j=0; j<n; j++)
      //  for(k=0; k<j; k++)
      //     G[j*n+k] = G[k*n+j];

      sgesvd_("n", "a", &n, &n, G, &n, S, U, &n, VT, &n, work, &lwork, &info);

      mins = 100.f, maxs = 0.f;
      for(k=0; k<n; k++){
        S[k] = sqrt(S[k]);

	if (S[k] < mins)  mins = S[k];
	if (S[k] > maxs)  maxs = S[k];
      }

      for(k=0; k<n;k++){
        vt = VT + k*n;
        for(j=0; j<n; j++)
          vt[j]*=S[j];
      } 
      sgeqrf_(&n, &n, VT, &n, tau, work, &lwork, &info);

      if (i==1)
        scopy_(&n2, VT, &one, R, &one);
      else
        strmm_("l", "u", "n", "n", &n, &n, &alpha, VT, &n, R, &n);
      
      strsm_("r", "u", "n", "n", &m, &n, &alpha, VT, &n, A, &m);    

      if (mins > 0.00001f) 
        cn = maxs/mins;

      fprintf(stderr, "\nIteration %d, cond num = %f just cpu \n", i, cn);
    } while (cn > 100.f);

    free(G);
    free(VT);
    free(S);
    free(work);
    free(tau);

}

//=============================================================================

void chol_qr_it_GPU(int m, int n, float *d_A, int lda, float *G, float *R, 
                 float *work, int lwork){
  int i = 0, k, j, info, n2 = n*n, one = 1,lworksvd=n*n;
    float *U, *S, *VT, *vt, *tau,*worksvd;
    float cn = 200.f, alpha = 1.f, zero = 0.f, mins, maxs;
    // cublasHandle_t handle;
    //cublasCreate(&handle);

    VT   = (float*)malloc(n * n * 4);
    S    = (float*)malloc(    n * 4);
    tau  = (float*)malloc(    n * 4);
    worksvd = (float*)malloc(lworksvd* 4);
    do {
        i++;
	
      // cublasSgemm(&handle,CUBLAS_OP_T,CUBLAS_OP_N, n, n, m, &alpha, *d_A, m, *d_A, m, &zero, *G, n);
      cublasSgemm('T','N', n, n, m, alpha, d_A, m, d_A, m, zero, G, n);

      hipblasGetVector(n*n,sizeof(float),G,1,work,1 );

      sgesvd_("n", "a", &n, &n, work, &n, S, U, &n, VT, &n, worksvd, &lworksvd, &info);
                  fprintf(stderr,"hi \n");    
      // sgesvd_( ... );

      mins = 100.f, maxs = 0.f;
      for(k=0; k<n; k++){
        S[k] = sqrt(S[k]);

        if (S[k] < mins)  mins = S[k];
        if (S[k] > maxs)  maxs = S[k];
      }

      for(k=0; k<n;k++){
        vt = VT + k*n;
        for(j=0; j<n; j++)
          vt[j]*=S[j];
      }
      sgeqrf_(&n, &n, VT, &n, tau, work, &lwork, &info);
      //      sgeqrf_( ... );

      if (i==1)
        scopy_(&n2, VT, &one, R, &one);	
      // scopy_( ... );
      else
	strmm_("l", "u", "n", "n", &n, &n, &alpha, VT, &n, R, &n);
      
      hipblasSetVector(n*n,sizeof(float),R,1,G,1 );
      cublasStrsm('r', 'u', 'N', 'N', m, n, alpha, G, n, d_A, m);    

      if (mins > 0.00001f)
        cn = maxs/mins;
      
      fprintf(stderr, "\nIteration %d, cond num = %f \n", i, cn);
       } while (cn > 100.f);
    //    cublasDestroy(&handle);
    free(VT);
    free(S);
    free(tau);
    free(worksvd);
    }

//=============================================================================
